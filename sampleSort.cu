#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <algorithm>

// Kernel for sorting chunks of the array
__global__ void sort_kernel(int* data, int chunk_size, int total_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int start = idx * chunk_size;
    int end = min(start + chunk_size, total_size);

    if (start < total_size) {
        // Sort the assigned chunk using Thrust
        thrust::sort(thrust::device, data + start, data + end);
    }
}

// Host function for parallel sample sort
void parallel_samplesort_cuda(int* data, int size, int num_chunks) {
    int chunk_size = size / num_chunks; // Calculate the size of each chunk
    int* d_data;

    // Allocate memory on the device
    hipMalloc(&d_data, size * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_data, data, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    sort_kernel<<<num_chunks, 256>>>(d_data, chunk_size, size);

    // Copy sorted data back from device to host
    hipMemcpy(data, d_data, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);
}

int main() {
    const int size = 1024; // Total size of the array
    const int num_chunks = 4; // Number of chunks to divide the array into
    int data[size];
    const int repetitions = 10; // Number of repetitions for averaging

    // Fill the array with random values
    for (int i = 0; i < size; i++) {
        data[i] = rand() % 1000; // Random values between 0 and 999
    }

    // Variables for timing
    double total_time = 0.0;

    // Run the sort multiple times and measure the time
    for (int i = 0; i < repetitions; i++) {
        int temp_data[size];
        std::copy(data, data + size, temp_data); // Copy original data for consistency

        auto start = std::chrono::high_resolution_clock::now();
        parallel_samplesort_cuda(temp_data, size, num_chunks);
        auto end = std::chrono::high_resolution_clock::now();

        std::chrono::duration<double, std::milli> elapsed = end - start;
        total_time += elapsed.count();
    }

    // Calculate the average time
    double average_time = total_time / repetitions;

   
    // Print the average time
    std::cout << "Array Size: " << size << std::endl;
    std::cout << "Number of Chunks: " << num_chunks << std::endl;
    std::cout << "Average Time: " << average_time << " ms" << std::endl;

    return 0;
}
